
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void sum_to_vector(float *A, float *B)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
    {
        B[index] = A[index] + N;
    }
}

int main()
{
    float h_A[N];
    float h_B[N];

    for (int i = 0; i < N; ++i)
    {
        h_A[i] = (float)(i + 1);
    }

    float *d_A;
    float *d_B;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 32;
    int gridSize = (N + blockSize - 1) / blockSize;
    sum_to_vector<<<gridSize, blockSize>>>(d_A, d_B);

    hipMemcpy(h_B, d_B, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant Vector:\n");
    for (int i = 0; i < N; ++i)
    {
        printf("%.2f\t", h_B[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
