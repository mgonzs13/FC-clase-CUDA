
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10

// CUDA kernel to multiply two matrices
__global__ void multiply_matrices(int *mat1, int *mat2, int *result)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N)
    {

        int sum = 0;
        for (int i = 0; i < N; ++i)
        {
            sum += mat1[row * N + i] * mat2[i * N + col];
        }
        result[row * N + col] = sum;
    }
}

// Function to initialize a matrix with random values between 5 and 15
void initializeMatrix(int *mat, int size)
{
    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            mat[i * size + j] = rand() % 11 + 5; // Values between 5 and 15
        }
    }
}

// Function to print a matrix
void printMatrix(int *mat, int size)
{
    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            printf("%d\t", mat[i * size + j]);
        }
        printf("\n");
    }
}

int main()
{

    // Host matrices and result
    int *h_mat1, *h_mat2, *h_result;
    size_t bytes = N * N * sizeof(int);

    h_mat1 = (int *)malloc(bytes);
    h_mat2 = (int *)malloc(bytes);
    h_result = (int *)malloc(bytes);

    // Initialize matrices with random values
    initializeMatrix(h_mat1, N);
    initializeMatrix(h_mat2, N);

    // Print matrices if needed
    // printf("Matrix 1:\n");
    // printMatrix(h_mat1, N);
    // printf("\nMatrix 2:\n");
    // printMatrix(h_mat2, N);

    // Device matrices and result
    int *d_mat1, *d_mat2, *d_result;
    hipMalloc((void **)&d_mat1, bytes);
    hipMalloc((void **)&d_mat2, bytes);
    hipMalloc((void **)&d_result, bytes);

    // Copy data from host to device
    hipMemcpy(d_mat1, h_mat1, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, h_mat2, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_result, h_result, bytes, hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 blockSize(N, N); // 16x16 thread block
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    multiply_matrices<<<gridSize, blockSize>>>(d_mat1, d_mat2, d_result);

    // Copy the result back to the host
    hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);

    // Print the result if needed
    printf("\nResult Matrix:\n");
    printMatrix(h_result, N);

    // Free device memory
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_result);

    // Free host memory
    free(h_mat1);
    free(h_mat2);
    free(h_result);

    return 0;
}
